#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    hipSetDevice(0);

    // メモリを適当に確保して解放
    const size_t size = 1 << 20;  // 1 MB
    void* d_ptr = nullptr;
    hipMalloc(&d_ptr, size);
    hipMemset(d_ptr, 0, size);
    hipFree(d_ptr);

    // 別ターミナルでnvidia-smiで確認
    std::cout << "Check nvidia-smi in another terminal. Press Enter to exit." << std::endl;
    std::cin.get();

    return 0;
}
